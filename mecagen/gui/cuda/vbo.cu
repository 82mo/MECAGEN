/*
 * Copyright (C) 2015 by Julien Delile
 * 
 * This file is part of MECAGEN.
 * 
 * MECAGEN is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation version 3 of the License, or
 * any later version.
 * 
 * MECAGEN is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with MECAGEN.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "vbo.h"

#include "kernel.cu"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>

// Cuda interface:
void allocateArray(void **devPtr, size_t size)
{ hipMalloc(devPtr, size); }

void freeArray(void *devPtr)
{ hipFree(devPtr); }

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{ hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice); }

void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{ hipMemcpy(host, device, size, hipMemcpyDeviceToHost); }

// OpenGL interoperability:
void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{ hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, hipGraphicsRegisterFlagsNone); }

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{ hipGraphicsUnregisterResource(cuda_vbo_resource); }

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
  // gpuErrchk(hipDeviceSynchronize());
  void *ptr;
  // gpuErrchk(hipDeviceSynchronize());
  hipGraphicsMapResources(1, cuda_vbo_resource, 0);
  // gpuErrchk(hipDeviceSynchronize());
  size_t num_bytes;
  // gpuErrchk(hipDeviceSynchronize());
  hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes, *cuda_vbo_resource);
  // gpuErrchk(hipDeviceSynchronize());
  return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{ hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0); }


void funcCudaGetLastError(){

	// int deviceCount;
	// hipGetDeviceCount(&deviceCount);
	// int device;
	// for (device = 0; device < deviceCount; ++device) {
	// hipDeviceProp_t deviceProp;
	// hipGetDeviceProperties(&deviceProp, device);
	// printf("Device %d has compute capability %d.%d.\n",
	// device, deviceProp.major, deviceProp.minor);
	// }

	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("CUDA error at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(error));
		exit(-1);
	}
}

void update_vbo(
    mg::DisplayParams *hostDisplayParams,
    float*      outputVBO,            // Where to write on the GPU
    uint        cellNum,                // number of cells
    mg::d3*     cellPosition,                // cell position
    mg::d3*     cellRadius,             // cell radius
    uint*       cellTopologicalNeighbNum,
    uint*       cellTopologicalNeighbId,
    mg::d3*    cellAxisAB,               // Cell axis
    float*      SphereVertex,           // Sphere vertex
    uint*       SphereIndice,           // Sphere indice
    uint        numNeighbMax,
    uint*       SphereVertexNumTriVois, // Voir d'où ça vient
    uint*       SphereVertexIDTriVois,  
    uint*       cellPopulation,
    double*      cellLigand,
    double*      cellProtein,
    mg::f3  colorPop0,
    mg::f3  colorPop1,
    mg::f3  colorPop2,
    mg::f3  colorPop3,
    mg::d3*  cellTriangleNormales,
    uint*       cellType,
    uint*       cellEpiIsPolarized,
    uint*       cellEpiId
    )
{
  
  hipMemcpyToSymbol<mg::DisplayParams>( displayParams, hostDisplayParams, sizeof(mg::DisplayParams));

  // gpuErrchk(hipDeviceSynchronize());
  
  update_vbo_old_gpu_1_D<<< cellNum, NUMSPHEREVERTEX >>>(
      outputVBO,            // Where to write on the GPU
        cellNum,                // number of cells
        cellPosition,                // cell position
        cellRadius,             // cell radius
        cellTopologicalNeighbNum,
        cellTopologicalNeighbId,
        cellAxisAB,               // Cell axis
        SphereVertex,           // Sphere vertex
        numNeighbMax,
        cellType,
        cellEpiIsPolarized
    );

  gpuErrchk(hipDeviceSynchronize());

  update_vbo_old_gpu_2_D<<< cellNum, NUMSPHERETRIANGLE >>>(
      outputVBO,            // Where to write on the GPU
        cellNum,                // number of cells
        SphereIndice,           // Sphere indice
        cellTriangleNormales
    );

  gpuErrchk(hipDeviceSynchronize());

	update_vbo_old_gpu_3_D<<< cellNum, NUMSPHERETRIANGLE >>>(
			outputVBO,            // Where to write on the GPU
		    cellNum,                // number of cells
    		SphereVertexNumTriVois,
    		SphereVertexIDTriVois,  
        cellPopulation,
    		cellLigand,
    		colorPop0,
    		colorPop1,
    		colorPop2,
    		colorPop3,
        cellTriangleNormales,
        cellProtein,
        cellType,
        cellEpiIsPolarized,
        cellEpiId
		);

	gpuErrchk(hipDeviceSynchronize());

  // float * temp = new float[10 * NUMSPHEREVERTEX *cellNum];
  // // allocateArray((void**)&temp, 10 * NUMSPHEREVERTEX *cellNum *sizeof(float));
  // copyArrayFromDevice(temp, outputVBO, 0, cellNum2 * 10 * NUMSPHEREVERTEX *sizeof(float));

  // for(uint i=0; i<NUMSPHEREVERTEX *cellNum2;i++){
  //   std::cout << "vertex " << i << " : "  
  //             << temp[10*i+0] << " " 
  //             << temp[10*i+1] << " " 
  //             << temp[10*i+2] << " " 
  //             << temp[10*i+3] << " " 
  //             << temp[10*i+4] << " " 
  //             << temp[10*i+5] << " " 
  //             << temp[10*i+6] << " " 
  //             << temp[10*i+7] << " " 
  //             << temp[10*i+8] << " " 
  //             << temp[10*i+9] << std::endl; 
  // }
  // delete [] temp;
 	
}